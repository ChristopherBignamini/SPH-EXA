#include "hip/hip_runtime.h"
#include <chrono>

#include "ryoanji/types.h"
#include "ryoanji/buildtree_cs.hpp"
#include "ryoanji/traversal_cpu.hpp"
#include "ryoanji/dataset.h"
#include "ryoanji/traversal.h"
#include "ryoanji/direct.cuh"
#include "ryoanji/upwardpass.h"

int main(int argc, char** argv)
{
    int power = argc > 1 ? std::stoi(argv[1]) : 17;
    int directRef = argc > 2 ? std::stoi(argv[2]) : 1;

    int numBodies = (1 << power) - 1;
    int images    = 0;
    float theta   = 0.6;
    float boxSize = 3;

    const float eps   = 0.05;
    const int ncrit   = 64;
    const float cycle = 2 * M_PI;

    fprintf(stdout, "--- BH Parameters ---------------\n");
    fprintf(stdout, "numBodies            : %d\n", numBodies);
    fprintf(stdout, "P                    : %d\n", P);
    fprintf(stdout, "theta                : %f\n", theta);
    fprintf(stdout, "ncrit                : %d\n", ncrit);

    auto bodies = makeCubeBodies(numBodies, boxSize);

    Box box{ {0.0f}, boxSize * 1.00f};

    cudaVec<CellData> sources(0, true);

    auto [highestLevel, levelRangeCs] = buildFromCstone(bodies, box, sources);

    int numSources = sources.size();

    cudaVec<fvec4> bodyPos(numBodies, true);
    std::copy(bodies.begin(), bodies.end(), bodyPos.h());
    bodyPos.h2d();

    cudaVec<int2> levelRange(levelRangeCs.size(), true);
    std::copy(levelRangeCs.begin(), levelRangeCs.end(), levelRange.h());
    levelRange.h2d();

    cudaVec<fvec4> sourceCenter(numSources, true);
    cudaVec<fvec4> Multipole(NVEC4 * numSources, true);

    int numLeaves = -1;
    Pass::upward(numLeaves, highestLevel, theta, levelRange, bodyPos, sources, sourceCenter, Multipole);
    sourceCenter.d2h();
    Multipole.d2h();

    cudaVec<fvec4> bodyAcc(numBodies, true);

    fprintf(stdout, "--- BH Profiling ----------------\n");

    auto t0 = std::chrono::high_resolution_clock::now();

    fvec4 interactions = Traversal::approx(0,
                                           bodyPos.size(),
                                           images,
                                           eps,
                                           cycle,
                                           bodyPos,
                                           bodyAcc,
                                           sources,
                                           sourceCenter,
                                           Multipole,
                                           levelRange);

    auto t1      = std::chrono::high_resolution_clock::now();
    double dt    = std::chrono::duration<double>(t1 - t0).count();
    double flops = (interactions[0] * 20 + interactions[2] * 2 * pow(P, 3)) * numBodies / dt / 1e12;

    fprintf(stdout, "--- Total runtime ----------------\n");
    fprintf(stdout, "Total BH            : %.7f s (%.7f TFlops)\n", dt, flops);

    if (!directRef) { return 0; }

    cudaVec<fvec4> bodyAccDirect(numBodies, true);

    t0 = std::chrono::high_resolution_clock::now();
    directSum(eps, bodyPos, bodyAccDirect);
    t1 = std::chrono::high_resolution_clock::now();
    dt = std::chrono::duration<double>(t1 - t0).count();

    flops = 24. * numBodies * numBodies / dt / 1e12;
    fprintf(stdout, "Total Direct         : %.7f s (%.7f TFlops)\n", dt, flops);

    bodyAcc.d2h();
    bodyAccDirect.d2h();

    std::vector<double> delta(numBodies);

    for (int i = 0; i < numBodies; i++)
    {
        fvec3 ref   = {bodyAccDirect[i][1], bodyAccDirect[i][2], bodyAccDirect[i][3]};
        fvec3 probe = {bodyAcc[i][1], bodyAcc[i][2], bodyAcc[i][3]};
        delta[i]    = std::sqrt(norm2(ref - probe) / norm2(ref));
    }

    //int mei = std::max_element(delta.begin(), delta.end()) - delta.begin();
    //fvec4 test = walkParticle(mei, eps, sources, sourceCenter, Multipole, bodyPos);
    //std::cout << bodyAcc[mei][1] << " " << test[1] << std::endl;

    std::sort(begin(delta), end(delta));

    fprintf(stdout, "--- BH vs. direct ---------------\n");

    std::cout << "min Error: "       << delta[0] << std::endl;
    std::cout << "50th percentile: " << delta[numBodies/2] << std::endl;
    std::cout << "10th percentile: " << delta[numBodies*0.9] << std::endl;
    std::cout << "1st percentile: "  << delta[numBodies*0.99] << std::endl;
    std::cout << "max Error: "       << delta[numBodies-1] << std::endl;

    //fprintf(stdout, "--- FMM vs. direct ---------------\n");
    //fprintf(stdout, "Rel. L2 Error (pot)  : %.7e\n", sqrt(diffp / normp));
    //fprintf(stdout, "Rel. L2 Error (acc)  : %.7e\n", sqrt(diffa / norma));

    fprintf(stdout, "--- Tree stats -------------------\n");
    fprintf(stdout, "Bodies               : %d\n", numBodies);
    fprintf(stdout, "Cells                : %d\n", numSources);
    fprintf(stdout, "Tree depth           : %d\n", 0);
    fprintf(stdout, "--- Traversal stats --------------\n");
    fprintf(stdout, "P2P mean list length : %d (max %d)\n", int(interactions[0]), int(interactions[1]));
    fprintf(stdout, "M2P mean list length : %d (max %d)\n", int(interactions[2]), int(interactions[3]));

    return 0;
}

